#include "SART_cuda.h" // consists all required package and functions

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
// Macro for input and output
#define IN_IMG prhs[0]
#define GEO_PARA prhs[1]
#define ITER_PARA prhs[2]
#define OUT_PROJ plhs[0]

int nx, ny, nz, na, nb, numImg, numBytesImg, numSingleProj, numBytesSingleProj;
float da, db, ai, bi, SO, SD;

// resolutions of volumes 
if (mxGetField(GEO_PARA, 0, "nx") != NULL)
    nx = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nx"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution nx.\n");

if (mxGetField(GEO_PARA, 0, "ny") != NULL)
    ny = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "ny"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution ny.\n");

if (mxGetField(GEO_PARA, 0, "nz") != NULL)
    nz = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nz"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution nz.\n");

numImg = nx * ny * nz; // size of image
numBytesImg = numImg * sizeof(float); // number of bytes in image

// detector plane resolutions
if (mxGetField(GEO_PARA, 0, "na") != NULL)
    na = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "na"));
else if (mxGetField(GEO_PARA, 0, "nv") != NULL)
    na = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nv"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number of detector in plane, which is denoted as na or nu.\n");

if (mxGetField(GEO_PARA, 0, "nb") != NULL)
    nb = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nb"));
else if (mxGetField(GEO_PARA, 0, "nu") != NULL)
    nb = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nu"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number of detector across plane, which is denoted as nb or nv.\n");

numSingleProj = na * nb;
numBytesSingleProj = numSingleProj * sizeof(float);

// detector resolution
if (mxGetField(GEO_PARA, 0, "da") != NULL)
    da = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "da"));
else{
    da = 1.0f;
    mexPrintf("Automatically set detector cell size da to 1. \n");
    mexPrintf("If don't want that default value, please set para.da manually.\n");
}

if (mxGetField(GEO_PARA, 0, "db") != NULL)
    db = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "db"));
else{
    db = 1.0f;
    mexPrintf("Automatically set detectof cell size db to 1. \n");
    mexPrintf("If don't want that default value, please set para.db manually.\n");
}


// detector plane offset from centered calibrations
if (mxGetField(GEO_PARA, 0, "ai") != NULL){
    ai = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "ai"));
    ai -= (float)na / 2 - 0.5f;
}
else{
    mexPrintf("Automatically set detector offset ai to 0. \n");
    mexPrintf("If don't want that default value, please set para.ai manually.\n");
    ai = - (float)na / 2 + 0.5f;
}

if (mxGetField(GEO_PARA, 0, "bi") != NULL){
    bi = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "bi"));
    if (bi > -1)
        bi -= (float)nb / 2 - 0.5f;
}
else{
    mexPrintf("Automatically set detector offset bi to 0. \n");
    mexPrintf("If don't want that default value, please set para.bi manually.\n");
    bi = - (float)nb / 2 + 0.5f;
}


if (mxGetField(GEO_PARA, 0, "SO") != NULL)
    SO = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SO"));
else if (mxGetField(GEO_PARA, 0, "SI") != NULL)
    SO = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SI"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid distance between source and isocenter, which is denoted with para.SO or para.DI.\n");

if (mxGetField(GEO_PARA, 0, "SD") != NULL)
    SD = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SD"));
else if (mxGetField(GEO_PARA, 0, "DI") != NULL)
    SD = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "DI")) + SO;
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid distance between source and detector plane, which is denoted with para.SD or para.SI + para.DI.\n");


// load iterating parameters, for the whole bin

int numProj, numBytesProj, N_view; // number of bins, numbers of views of bins, and the index view of each bin.
// e.g. we have 3 bins here with 10 total views. For these 3 bins, they holds 1,3,6 views. Then we will set n_views as {0, 1, 4, 10}, which is the starting view indices of each bin. Moreover, we need to well arrange the volumes and flows.


if (mxGetField(ITER_PARA, 0, "N_views") != NULL)
    N_view = (int)mxGetScalar(mxGetField(ITER_PARA, 0, "N_views"));
else{
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number angles, which is denoted as iter_para.N_views.\n");
}

// 5D models
float *h_alpha_x, *h_alpha_y, *h_alpha_z, *h_beta_x, *h_beta_y, *h_beta_z, *angles;

// load 5DCT alpha and beta
if (mxGetField(ITER_PARA, 0, "alpha_x") != NULL)
    h_alpha_x = (float*)mxGetData(mxGetField(ITER_PARA, 0, "alpha_x")); 
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.alpha_x.\n");    

if (mxGetField(ITER_PARA, 0, "alpha_y") != NULL)
    h_alpha_y = (float*)mxGetData(mxGetField(ITER_PARA, 0, "alpha_y")); 
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.alpha_y.\n");

if (mxGetField(ITER_PARA, 0, "alpha_z") != NULL)
    h_alpha_z = (float*)mxGetData(mxGetField(ITER_PARA, 0, "alpha_z"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.alpha_z.\n");

if (mxGetField(ITER_PARA, 0, "beta_x") != NULL)
    h_beta_x = (float*)mxGetData(mxGetField(ITER_PARA, 0, "beta_x"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.beta_x.\n");

if (mxGetField(ITER_PARA, 0, "beta_y") != NULL)
    h_beta_y = (float*)mxGetData(mxGetField(ITER_PARA, 0, "beta_y")); 
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.beta_y.\n");

if (mxGetField(ITER_PARA, 0, "beta_z") != NULL)
    h_beta_z = (float*)mxGetData(mxGetField(ITER_PARA, 0, "beta_z"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.beta_z.\n");

// load 5DCT parameters volume (v) and flow (f)
float *volumes, *flows;
if (mxGetField(ITER_PARA, 0, "volumes") != NULL)
    volumes= (float*)mxGetData(mxGetField(ITER_PARA, 0, "volumes"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume in iter_para.volumes.\n");  

if (mxGetField(ITER_PARA, 0, "flows") != NULL)
    flows = (float*)mxGetData(mxGetField(ITER_PARA, 0, "flows"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid flow in iter_para.flows.\n");    

if (mxGetField(ITER_PARA, 0, "angles") != NULL)
    angles = (float*)mxGetData(mxGetField(ITER_PARA, 0, "angles"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid angles iter_para.angles.\n");

numProj = numSingleProj * N_view;
numBytesProj = numProj * sizeof(float);

// load initial guess of image
float *h_img;
h_img = (float*)mxGetData(IN_IMG);


// define thread distributions
const dim3 gridSize_img((nx + BLOCKWIDTH - 1) / BLOCKWIDTH, (ny + BLOCKHEIGHT - 1) / BLOCKHEIGHT, (nz + BLOCKDEPTH - 1) / BLOCKDEPTH);
const dim3 gridSize_singleProj((nb + BLOCKWIDTH - 1) / BLOCKWIDTH, (na + BLOCKHEIGHT - 1) / BLOCKHEIGHT, 1);
const dim3 blockSize(BLOCKWIDTH,BLOCKHEIGHT, BLOCKDEPTH);

// CUDA 3DArray Malloc parameters
struct hipExtent extent_img = make_hipExtent(nx, ny, nz);
struct hipExtent extent_singleProj = make_hipExtent(nb, na, 1);

//Allocate CUDA array in device memory of 5DCT matrices: alpha and beta
hipArray *d_alpha_x, *d_alpha_y, *d_alpha_z, *d_beta_x, *d_beta_y, *d_beta_z;
hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

hipError_t cudaStat;
// alpha_x
cudaStat = hipMalloc3DArray(&d_alpha_x, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for alpha_x failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

// alpha_y
cudaStat = hipMalloc3DArray(&d_alpha_y, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for alpha_y failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

// alpha_z
cudaStat = hipMalloc3DArray(&d_alpha_z, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for alpha_z failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

// beta_x
cudaStat = hipMalloc3DArray(&d_beta_x, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for beta_x failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}
// beta_y
cudaStat = hipMalloc3DArray(&d_beta_y, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for beta_y failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}
// beta_z
cudaStat = hipMalloc3DArray(&d_beta_z, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for beta_z failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}


// Get pitched pointer to alpha and beta in host memory
hipPitchedPtr hp_alpha_x = make_hipPitchedPtr((void*) h_alpha_x, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_alpha_y = make_hipPitchedPtr((void*) h_alpha_y, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_alpha_z = make_hipPitchedPtr((void*) h_alpha_z, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_beta_x = make_hipPitchedPtr((void*) h_beta_x, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_beta_y = make_hipPitchedPtr((void*) h_beta_y, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_beta_z = make_hipPitchedPtr((void*) h_beta_z, nx * sizeof(float), nx, ny);

// Copy alpha and beta to texture memory from pitched pointer
hipMemcpy3DParms copyParams = {0};
copyParams.extent = extent_img;
copyParams.kind = hipMemcpyHostToDevice;

//alpha_x
copyParams.srcPtr = hp_alpha_x;
copyParams.dstArray = d_alpha_x;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy alpha_x to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//alpha_y
copyParams.srcPtr = hp_alpha_y;
copyParams.dstArray = d_alpha_y;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy alpha_y to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//alpha_z
copyParams.srcPtr = hp_alpha_z;
copyParams.dstArray = d_alpha_z;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy alpha_z to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//beta_x
copyParams.srcPtr = hp_beta_x;
copyParams.dstArray = d_beta_x;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy beta_x to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//beta_y
copyParams.srcPtr = hp_beta_y;
copyParams.dstArray = d_beta_y;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy beta_y to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//beta_z
copyParams.srcPtr = hp_beta_z;
copyParams.dstArray = d_beta_z;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy beta_z to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}


// create texture object alpha and beta
hipResourceDesc resDesc;
hipTextureDesc texDesc, texDesc2;
memset(&resDesc, 0, sizeof(resDesc));
resDesc.resType = hipResourceTypeArray;

memset(&texDesc, 0, sizeof(texDesc));
texDesc.addressMode[0] = hipAddressModeClamp;
texDesc.addressMode[1] = hipAddressModeClamp;
texDesc.addressMode[2] = hipAddressModeClamp;
texDesc.filterMode = hipFilterModeLinear;
texDesc.readMode = hipReadModeElementType;
texDesc.normalizedCoords = 0;

memset(&texDesc2, 0, sizeof(texDesc2));
texDesc2.addressMode[0] = hipAddressModeClamp;
texDesc2.addressMode[1] = hipAddressModeClamp;
texDesc2.addressMode[2] = hipAddressModeClamp;
texDesc2.filterMode = hipFilterModePoint;
texDesc2.readMode = hipReadModeElementType;
texDesc2.normalizedCoords = 0;

// alpha_x
resDesc.res.array.array = d_alpha_x;
hipTextureObject_t tex_alpha_x = 0;
hipCreateTextureObject(&tex_alpha_x, &resDesc, &texDesc, NULL);

// alpha_y
resDesc.res.array.array = d_alpha_y;
// memset(&texDesc, 0, sizeof(texDesc));
// texDesc.addressMode[0] = hipAddressModeClamp;
// texDesc.addressMode[1] = hipAddressModeClamp;
// texDesc.addressMode[2] = hipAddressModeClamp;
// texDesc.filterMode = hipFilterModeLinear;
// texDesc.readMode = hipReadModeElementType;
// texDesc.normalizedCoords = 0;
hipTextureObject_t tex_alpha_y = 0;
hipCreateTextureObject(&tex_alpha_y, &resDesc, &texDesc, NULL);

// alpha_z
resDesc.res.array.array = d_alpha_z;
// memset(&texDesc, 0, sizeof(texDesc));
// texDesc.addressMode[0] = hipAddressModeClamp;
// texDesc.addressMode[1] = hipAddressModeClamp;
// texDesc.addressMode[2] = hipAddressModeClamp;
// texDesc.filterMode = hipFilterModeLinear;
// texDesc.readMode = hipReadModeElementType;
// texDesc.normalizedCoords = 0;
hipTextureObject_t tex_alpha_z = 0;
hipCreateTextureObject(&tex_alpha_z, &resDesc, &texDesc, NULL);

// beta_x
resDesc.res.array.array = d_beta_x;
// memset(&texDesc, 0, sizeof(texDesc));
// texDesc.addressMode[0] = hipAddressModeClamp;
// texDesc.addressMode[1] = hipAddressModeClamp;
// texDesc.addressMode[2] = hipAddressModeClamp;
// texDesc.filterMode = hipFilterModeLinear;
// texDesc.readMode = hipReadModeElementType;
// texDesc.normalizedCoords = 0;
hipTextureObject_t tex_beta_x = 0;
hipCreateTextureObject(&tex_beta_x, &resDesc, &texDesc, NULL);

// beta_y
resDesc.res.array.array = d_beta_y;
// memset(&texDesc, 0, sizeof(texDesc));
// texDesc.addressMode[0] = hipAddressModeClamp;
// texDesc.addressMode[1] = hipAddressModeClamp;
// texDesc.addressMode[2] = hipAddressModeClamp;
// texDesc.filterMode = hipFilterModeLinear;
// texDesc.readMode = hipReadModeElementType;
// texDesc.normalizedCoords = 0;
hipTextureObject_t tex_beta_y = 0;
hipCreateTextureObject(&tex_beta_y, &resDesc, &texDesc, NULL);

// beta_z
resDesc.res.array.array = d_beta_z;
// memset(&texDesc, 0, sizeof(texDesc));
// texDesc.addressMode[0] = hipAddressModeClamp;
// texDesc.addressMode[1] = hipAddressModeClamp;
// texDesc.addressMode[2] = hipAddressModeClamp;
// texDesc.filterMode = hipFilterModeLinear;
// texDesc.readMode = hipReadModeElementType;
// texDesc.normalizedCoords = 0;
hipTextureObject_t tex_beta_z = 0;
hipCreateTextureObject(&tex_beta_z, &resDesc, &texDesc, NULL);

// malloc in device: projection of the whole bin
float *d_proj;
hipMalloc((void**)&d_proj, numBytesSingleProj);

// malloc in device: projection of the whole bin
float *d_img ,*d_img1;
hipArray* array_img;
hipMalloc((void**)&d_img, numBytesImg);
hipMalloc((void**)&d_img1, numBytesImg);
hipMemcpy(d_img, h_img, numBytesImg, hipMemcpyHostToDevice);
cudaStat = hipMalloc3DArray(&array_img, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_img failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

float angle, volume, flow;

//Malloc forward and inverted DVFs in device
float *d_mx, *d_my, *d_mz, *d_mx2, *d_my2, *d_mz2;
hipMalloc(&d_mx, numBytesImg);
hipMalloc(&d_my, numBytesImg);
hipMalloc(&d_mz, numBytesImg);
hipMalloc(&d_mx2, numBytesImg);
hipMalloc(&d_my2, numBytesImg);
hipMalloc(&d_mz2, numBytesImg);


// Alloc forward and inverted DVFs in device, in form of array memory
hipArray *array_mx, *array_my, *array_mz, *array_mx2, *array_my2, *array_mz2;
cudaStat = hipMalloc3DArray(&array_mx, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_mx failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

cudaStat = hipMalloc3DArray(&array_my, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_my failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

cudaStat = hipMalloc3DArray(&array_mz, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_mz failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

cudaStat = hipMalloc3DArray(&array_mx2, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_mx2 failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

cudaStat = hipMalloc3DArray(&array_my2, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_my2 failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

cudaStat = hipMalloc3DArray(&array_mz2, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_mz2 failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

// define tex_mx etc
hipTextureObject_t tex_mx = 0, tex_my = 0, tex_mz = 0, tex_mx2 = 0, tex_my2 = 0, tex_mz2 = 0, tex_img = 0;


// setup output images
OUT_PROJ = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS, mxREAL);
const mwSize outDim[3] = {(mwSize)nb, (mwSize)na, (mwSize)N_view};

mxSetDimensions(OUT_PROJ, outDim, 3);
mxSetData(OUT_PROJ, mxMalloc(numBytesProj));
float *h_outproj = (float*)mxGetData(OUT_PROJ);
copyParams.kind = hipMemcpyDeviceToDevice;

hipPitchedPtr dp_img = make_hipPitchedPtr((void*) d_img, nx * sizeof(float), nx, ny);
copyParams.srcPtr = dp_img;
copyParams.dstArray = array_img;
cudaStat = hipMemcpy3D(&copyParams);   
if (cudaStat != hipSuccess) {
    mexPrintf("Failed to copy dp_img to array memory array_img.\n");
    mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}
resDesc.res.array.array = array_img;
hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);


for (int i_view = 0; i_view < N_view; i_view++)
{
    mexPrintf("Projecting %d view, of all %d views.\n", i_view, N_view); mexEvalString("drawnow;");
    volume = volumes[i_view];
    flow = flows[i_view];
    angle = angles[i_view];

    kernel_forwardDVF<<<gridSize_img, blockSize>>>(d_mx, d_my, d_mz, tex_alpha_x, tex_alpha_y, tex_alpha_z, tex_beta_x, tex_beta_y, tex_beta_z, volume, flow, nx, ny, nz);
    hipDeviceSynchronize();

    // copy mx etc to pitched pointer and bind it to a texture object
    hipPitchedPtr dp_mx = make_hipPitchedPtr((void*) d_mx, nx * sizeof(float), nx, ny);
    copyParams.srcPtr = dp_mx;
    copyParams.dstArray = array_mx;
    cudaStat = hipMemcpy3D(&copyParams);   
    if (cudaStat != hipSuccess) {
        mexPrintf("Failed to copy dp_mx to array memory array_mx2.\n");
        mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
            mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
    }
    resDesc.res.array.array = array_mx;
    hipCreateTextureObject(&tex_mx, &resDesc, &texDesc, NULL);

    hipPitchedPtr dp_my = make_hipPitchedPtr((void*) d_my, nx * sizeof(float), nx, ny);
    copyParams.srcPtr = dp_my;
    copyParams.dstArray = array_my;
    cudaStat = hipMemcpy3D(&copyParams);   
    if (cudaStat != hipSuccess) {
        mexPrintf("Failed to copy dp_my to array memory array_mx2.\n");
        mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
            mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
    }
    resDesc.res.array.array = array_my;
    hipCreateTextureObject(&tex_my, &resDesc, &texDesc, NULL);

    hipPitchedPtr dp_mz = make_hipPitchedPtr((void*) d_mz, nx * sizeof(float), nx, ny);
    copyParams.srcPtr = dp_mz;
    copyParams.dstArray = array_mz;
    cudaStat = hipMemcpy3D(&copyParams);   
    if (cudaStat != hipSuccess) {
        mexPrintf("Failed to copy dp_mz to array memory array_mx2.\n");
        mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
            mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
    }
    resDesc.res.array.array = array_mz;
    hipCreateTextureObject(&tex_mz, &resDesc, &texDesc, NULL);

    kernel_invertDVF<<<gridSize_img, blockSize>>>(d_mx2, d_my2, d_mz2, tex_mx, tex_my, tex_mz, nx, ny, nz, 10);
    hipDeviceSynchronize();        
            
    kernel_deformation<<<gridSize_img, blockSize>>>(d_img1, tex_img, d_mx2, d_my2, d_mz2, nx, ny, nz);
    hipDeviceSynchronize();

    kernel_projection<<<gridSize_singleProj, blockSize>>>(d_proj, d_img1, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
    hipDeviceSynchronize();
    hipMemcpy(h_outproj + i_view * numSingleProj, d_proj, numBytesSingleProj, hipMemcpyDeviceToHost);
}


hipDestroyTextureObject(tex_alpha_x);
hipDestroyTextureObject(tex_alpha_y);
hipDestroyTextureObject(tex_alpha_z);
hipDestroyTextureObject(tex_beta_x);
hipDestroyTextureObject(tex_beta_y);
hipDestroyTextureObject(tex_beta_z);
hipDestroyTextureObject(tex_img);
hipDestroyTextureObject(tex_mx);
hipDestroyTextureObject(tex_my);
hipDestroyTextureObject(tex_mz);
hipDestroyTextureObject(tex_mx2);
hipDestroyTextureObject(tex_my2);
hipDestroyTextureObject(tex_mz2);

hipFreeArray(d_alpha_x);
hipFreeArray(d_alpha_y);
hipFreeArray(d_alpha_z);
hipFreeArray(d_beta_x);
hipFreeArray(d_beta_y);
hipFreeArray(d_beta_z);
// hipFreeArray(d_img);
hipFree(d_mx);
hipFree(d_my);
hipFree(d_mz);
hipFree(d_mx2);
hipFree(d_my2);
hipFree(d_mz2);
hipFreeArray(array_mx);
hipFreeArray(array_my);
hipFreeArray(array_mz);
hipFreeArray(array_mx2);
hipFreeArray(array_my2);
hipFreeArray(array_mz2);
hipFree(d_proj);

hipFree(d_img);
hipFree(d_img1);
hipDeviceReset();
return;
}

