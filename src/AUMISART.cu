#include "AUMISART.h" // consists all required package and functions

__host__ void host_AUMISART(float *h_outimg, float *h_outnorm, float *h_img, float *h_proj, int nx, int ny, int nz, int na, int nb, int outIter, int n_views, int n_iter, float da, float db, float ai, float bi, float SO, float SD, float dx, float lambda, float* volumes, float* flows, float* err_weights, float* angles)
{
    float *d_img, *d_img0, *d_img_temp, *d_proj, *d_proj_temp, *d_img_ones, *d_proj_ones;
    int numBytesImg = nx * ny * nz * sizeof(float);
    int numBytesProj = na * nb * sizeof(float);
    hipMalloc((void**)&d_img, numBytesImg);
    hipMalloc((void**)&d_img0, numBytesImg);
    hipMalloc((void**)&d_img_temp, numBytesImg);
    hipMalloc((void**)&d_img_ones, numBytesImg);
    hipMalloc((void**)&d_proj, numBytesProj);
    hipMalloc((void**)&d_proj_temp, numBytesProj);
    hipMalloc((void**)&d_proj_ones, numBytesProj);

    float *d_alpha_x, *d_alpha_y, *d_alpha_z, *d_beta_x, *d_beta_y, *d_beta_z;
    hipMalloc((void**)&d_alpha_x, numBytesImg);
    hipMalloc((void**)&d_alpha_y, numBytesImg);
    hipMalloc((void**)&d_alpha_z, numBytesImg);
    hipMalloc((void**)&d_beta_x, numBytesImg);
    hipMalloc((void**)&d_beta_y, numBytesImg);
    hipMalloc((void**)&d_beta_z, numBytesImg);

    host_initial(d_img, nx, ny, nz, 0.0f);
    host_initial(d_alpha_x, nx, ny, nz, 0.0f);
    host_initial(d_alpha_y, nx, ny, nz, 0.0f);
    host_initial(d_alpha_z, nx, ny, nz, 0.0f);
    host_initial(d_beta_x, nx, ny, nz, 0.0f);
    host_initial(d_beta_y, nx, ny, nz, 0.0f);
    host_initial(d_beta_z, nx, ny, nz, 0.0f);
    mexPrintf("Start iteration\n");
    mexPrintf("n_iter = %d\n", n_iter);
    mexPrintf("n_view = %d\n", n_views);
    //mexPrintf("Start iteration\n");
    
    for (int iter = 0; iter < n_iter; iter ++)
    {
        for (int i_view = 0; i_view < n_views; i_view ++)
        {   
            processBar(i_view, n_views, iter, n_iter);
            float volume_diff, flow_diff;
            if (i_view > 0)
            {
                volume_diff = volumes[i_view] - volumes[i_view - 1];
                flow_diff = flows[i_view] - flows[i_view - 1];
            
                host_deform(d_img_temp, d_img, nx, ny, nz, volume_diff, flow_diff, d_alpha_x, d_alpha_y, d_alpha_z, d_beta_x, d_beta_y, d_beta_z);
                hipMemcpy(d_img, d_img_temp, numBytesImg, hipMemcpyDeviceToDevice);
            }
            
            host_projection(d_proj_temp, d_img, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
            hipMemcpy(d_proj, h_proj + na * nb * i_view, numBytesProj, hipMemcpyHostToDevice);

            host_add(d_proj, d_proj_temp, nx, ny, nz, -1.0);

            host_backprojection(d_img_temp, d_proj, angles[i_view], SO, SD, na, nb, da, db, ai, bi, nx, ny, nz);

            host_initial(d_img_ones, nx, ny, nz, 1.0f);
            host_projection(d_proj_ones, d_img_ones, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
            host_backprojection(d_img_ones, d_proj_ones, angles[i_view], SO, SD, na, nb, da, db, ai, bi, nx, ny, nz);

            host_division(d_img_temp, d_img_ones, nx, ny, nz);

            host_add(d_img, d_img_temp, nx, ny, nz, lambda);

            if (i_view > 0)
            {
                host_update_udvf(d_alpha_x, d_alpha_y, d_alpha_z, d_beta_x, d_beta_y, d_beta_z, d_img, d_img0, volume_diff, flow_diff, nx, ny, nz, i_view);
            }
            hipMemcpy(d_img0, d_img, numBytesImg, hipMemcpyDeviceToDevice);    
        }
    }
    hipMemcpy(h_outimg, d_img, numBytesImg, hipMemcpyDeviceToHost);
    hipFree(d_img);
    hipFree(d_img0);
    hipFree(d_img_temp);
    hipFree(d_proj);
    hipFree(d_proj_temp);
    hipFree(d_img_ones);
    hipFree(d_proj_ones);
    hipFree(d_alpha_x);
    hipFree(d_alpha_y);
    hipFree(d_alpha_z);
    hipFree(d_beta_x);
    hipFree(d_beta_y);
    hipFree(d_beta_z);
    hipDeviceReset();
}
