#include "AUMISART.h" // consists all required package and functions

__host__ void host_AUMISART(float *h_outimg, float *h_outnorm, float *h_outalphax, float *h_img, float *h_proj, int nx, int ny, int nz, int na, int nb, int outIter, int n_views, int n_iter, int *op_iter, float da, float db, float ai, float bi, float SO, float SD, float dx, float lambda, float* volumes, float* flows, float* err_weights, float* angles)
{
    float *d_img, *d_img0, *d_img_temp, *d_proj, *d_proj_temp, *d_img_ones, *d_proj_ones;
    int numBytesImg = nx * ny * nz * sizeof(float);
    int numBytesProj = na * nb * sizeof(float);
    hipMalloc((void**)&d_img, numBytesImg);
    hipMalloc((void**)&d_img0, numBytesImg);
    hipMalloc((void**)&d_img_temp, numBytesImg);
    hipMalloc((void**)&d_img_ones, numBytesImg);
    hipMalloc((void**)&d_proj, numBytesProj);
    hipMalloc((void**)&d_proj_temp, numBytesProj);
    hipMalloc((void**)&d_proj_ones, numBytesProj);

    float *d_alpha_x, *d_alpha_y, *d_alpha_z, *d_beta_x, *d_beta_y, *d_beta_z;
    hipMalloc((void**)&d_alpha_x, numBytesImg);
    hipMalloc((void**)&d_alpha_y, numBytesImg);
    hipMalloc((void**)&d_alpha_z, numBytesImg);
    hipMalloc((void**)&d_beta_x, numBytesImg);
    hipMalloc((void**)&d_beta_y, numBytesImg);
    hipMalloc((void**)&d_beta_z, numBytesImg);
    hipMemcpy(d_img, h_img, numBytesImg, hipMemcpyHostToDevice);
    // host_initial(d_img, nx, ny, nz, 0.0f);
    host_initial(d_alpha_x, nx, ny, nz, 0.0f);
    host_initial(d_alpha_y, nx, ny, nz, 0.0f);
    host_initial(d_alpha_z, nx, ny, nz, 0.0f);
    host_initial(d_beta_x, nx, ny, nz, 0.0f);
    host_initial(d_beta_y, nx, ny, nz, 0.0f);
    host_initial(d_beta_z, nx, ny, nz, 0.0f);
    mexPrintf("Start iteration\n");

    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle); 
    float tempNorm, tempNorm0;

    for (int i_iter = 0; i_iter < n_iter; i_iter ++)
    {
        if (op_iter[i_iter] == 1)
        {   
            for (int i_view = 0; i_view < n_views; i_view ++)
            {   
                mexPrintf("iIter = %d / %d, and iView = %d / %d.\n", i_iter + 1, n_iter, i_view + 1, n_views); mexEvalString("drawnow;");

                host_projection(d_proj_temp, d_img, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
                hipMemcpy(d_proj, h_proj + na * nb * i_view, numBytesProj, hipMemcpyHostToDevice);
                stat = hipblasSnrm2(handle, na * nb, d_proj, 1, &tempNorm0);

                host_add(d_proj, d_proj_temp, na, nb, 1, -1.0);
                stat = hipblasSnrm2(handle, na * nb, d_proj, 1, &tempNorm);
                h_outnorm[i_iter * n_views + i_view] = tempNorm / tempNorm0;
            
                host_backprojection(d_img_temp, d_proj, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);

                host_initial(d_img_ones, nx, ny, nz, 1.0f);
                host_projection(d_proj_ones, d_img_ones, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
                host_backprojection(d_img_ones, d_proj_ones, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);

                host_division(d_img_temp, d_img_ones, nx, ny, nz);

                host_add(d_img, d_img_temp, nx, ny, nz, lambda);
            }
            hipMemcpy(h_outimg, d_img, numBytesImg, hipMemcpyDeviceToHost);
        }
        else
        {   
            if (i_iter == 0)
                hipMemcpy(d_img, h_img, numBytesImg, hipMemcpyHostToDevice);
            else
                hipMemcpy(d_img, h_outimg, numBytesImg, hipMemcpyHostToDevice);   
            for (int i_view = 1; i_view < n_views; i_view ++)
            {   
                mexPrintf("iIter = %d / %d, and iView = %d / %d.", i_iter + 1, n_iter, i_view + 1, n_views); 

                host_projection(d_proj_temp, d_img, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
                hipMemcpy(d_proj, h_proj + na * nb * i_view, numBytesProj, hipMemcpyHostToDevice);
                stat = hipblasSnrm2(handle, na * nb, d_proj, 1, &tempNorm0);
                float vd = volumes[i_view] - volumes[i_view - 1];
                float fd = flows[i_view] - flows[i_view - 1];
                host_add(d_proj, d_proj_temp, na, nb, 1, -1.0f); // new b
                host_initial(d_img0, nx, ny, nz, 0.0f);
                host_add2(d_img0, d_alpha_y, nx, ny, nz, d_img, vd, 1);
                host_add2(d_img0, d_alpha_x, nx, ny, nz, d_img, vd, 2);
                host_add2(d_img0, d_alpha_z, nx, ny, nz, d_img, vd, 3);
                host_add2(d_img0, d_beta_y, nx, ny, nz, d_img, fd, 1);
                host_add2(d_img0, d_beta_x, nx, ny, nz, d_img, fd, 2);
                host_add2(d_img0, d_beta_z, nx, ny, nz, d_img, fd, 3);
                host_projection(d_proj_temp, d_img0, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
                host_add(d_proj, d_proj_temp, na, nb, 1, 1.0f); // new b

                stat = hipblasSnrm2(handle, na * nb, d_proj, 1, &tempNorm);
                h_outnorm[i_iter * n_views + i_view] = tempNorm / tempNorm0;
                mexPrintf("error on projection = %f\n", tempNorm / tempNorm0);mexEvalString("drawnow;");
                host_backprojection(d_img_temp, d_proj, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);

                host_initial2(d_img_ones, nx, ny, nz, d_img, -vd, -fd);
                host_projection(d_proj_ones, d_img_ones, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
                host_backprojection(d_img_ones, d_proj_ones, angles[i_view], SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
                host_division(d_img_temp, d_img_ones, nx, ny, nz);

                host_add2(d_alpha_y, d_img_temp, nx, ny, nz, d_img, volumes[i_view - 1] - volumes[i_view], 1);
                host_add2(d_alpha_x, d_img_temp, nx, ny, nz, d_img, volumes[i_view - 1] - volumes[i_view], 2);
                host_add2(d_alpha_z, d_img_temp, nx, ny, nz, d_img, volumes[i_view - 1] - volumes[i_view], 3);
                host_add2(d_beta_y, d_img_temp, nx, ny, nz, d_img, flows[i_view - 1] - flows[i_view], 1);
                host_add2(d_beta_x, d_img_temp, nx, ny, nz, d_img, flows[i_view - 1] - flows[i_view], 2);
                host_add2(d_beta_z, d_img_temp, nx, ny, nz, d_img, flows[i_view - 1] - flows[i_view], 3);

                // hipMemcpy(d_img0, d_img, numBytesImg, hipMemcpyDeviceToDevice);
                // host_add2(d_img, d_alpha_x, nx, ny, nz, d_img0, volumes[i_view - 1] - volumes[i_view], 1);
                // host_add2(d_img, d_alpha_y, nx, ny, nz, d_img0, volumes[i_view - 1] - volumes[i_view], 2);                
                // host_add2(d_img, d_alpha_z, nx, ny, nz, d_img0, volumes[i_view - 1] - volumes[i_view], 3);                
                // host_add2(d_img, d_beta_x, nx, ny, nz, d_img0, flows[i_view - 1] - flows[i_view], 1);
                // host_add2(d_img, d_beta_y, nx, ny, nz, d_img0, flows[i_view - 1] - flows[i_view], 2);                
                // host_add2(d_img, d_beta_z, nx, ny, nz, d_img0, flows[i_view - 1] - flows[i_view], 3);  
                // break;
            }

        }
    }
    hipMemcpy(h_outalphax, d_alpha_x, numBytesImg, hipMemcpyDeviceToHost);
            
    hipMemcpy(h_outimg, d_img, numBytesImg, hipMemcpyDeviceToHost);   

    hipFree(d_img);
    hipFree(d_img);
    hipFree(d_img_temp);
    hipFree(d_proj);
    hipFree(d_proj_temp);
    hipFree(d_img_ones);
    hipFree(d_proj_ones);
    hipFree(d_alpha_x);
    hipFree(d_alpha_y);
    hipFree(d_alpha_z);
    hipFree(d_beta_x);
    hipFree(d_beta_y);
    hipFree(d_beta_z);
    hipblasDestroy(handle);

    hipDeviceReset();
}
