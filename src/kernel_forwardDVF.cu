
#include <hip/hip_runtime.h>
__global__ void kernel_forwardDVF(float *mx, float *my, float *mz, hipTextureObject_t alpha_x, hipTextureObject_t alpha_y, hipTextureObject_t alpha_z, hipTextureObject_t beta_x, hipTextureObject_t beta_y, hipTextureObject_t beta_z, float volume, float flow, int nx, int ny, int nz)
{
    int ix = 16 * blockIdx.x + threadIdx.x;
    int iy = 16 * blockIdx.y + threadIdx.y;
    int iz = 4 * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = ix + iy * nx + iz * nx * ny;    
    mx[id] = tex3D<float>(alpha_x, (ix + 0.5f), (iy + 0.5f), (iz + 0.5f)) * volume
           + tex3D<float>(beta_x, (ix + 0.5f), (iy + 0.5f), (iz + 0.5f)) * flow;
    my[id] = tex3D<float>(alpha_y, (ix + 0.5f), (iy + 0.5f), (iz + 0.5f)) * volume
           + tex3D<float>(beta_y, (ix + 0.5f), (iy + 0.5f), (iz + 0.5f)) * flow;
    mz[id] = tex3D<float>(alpha_z, (ix + 0.5f), (iy + 0.5f), (iz + 0.5f)) * volume
           + tex3D<float>(beta_z, (ix + 0.5f), (iy + 0.5f), (iz + 0.5f)) * flow;
}