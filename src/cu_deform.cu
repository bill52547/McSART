#include "hip/hip_runtime.h"
#include "cu_deform.h"

__host__ void host_deform(float *d_img1, float *d_img, int nx, int ny, int nz, float volume, float flow, float *alpha_x, float *alpha_y, float *alpha_z, float *beta_x, float *beta_y, float *beta_z)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, (nz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
    float *mx, *my, *mz;
    hipMalloc((void**)&mx, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&my, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&mz, nx * ny * nz * sizeof(float));
    kernel_forwardDVF<<<gridSize, blockSize>>>(mx, my, mz, alpha_x, alpha_y, alpha_z, beta_x, beta_y, beta_z, volume, flow, nx, ny, nz);
    hipDeviceSynchronize();
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipPitchedPtr dp_img = make_hipPitchedPtr((void*) d_img, nx * sizeof(float), nx, ny);
    hipMemcpy3DParms copyParams = {0};
    struct hipExtent extent_img = make_hipExtent(nx, ny, nz);
    copyParams.extent = extent_img;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = dp_img;
    hipArray *array_img;
    hipMalloc3DArray(&array_img, &channelDesc, extent_img);
    copyParams.dstArray = array_img;
    hipMemcpy3D(&copyParams);   

    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    resDesc.res.array.array = array_img;
    hipTextureObject_t tex_img = 0;
    hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);
    kernel_deformation<<<gridSize, blockSize>>>(d_img1, tex_img, mx, my, mz, nx, ny, nz);
    hipDeviceSynchronize();
    hipFree(mx);   
    hipFree(my);   
    hipFree(mz);   
    hipDestroyTextureObject(tex_img);
    hipFreeArray(array_img);

}

__host__ void host_deform2(float *d_img1, float *d_img, int nx, int ny, int nz, float volume, float flow, float *alpha_x, float *alpha_y, float *alpha_z, float *beta_x, float *beta_y, float *beta_z)
{
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, (nz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
    float *mx, *my, *mz;
    hipMalloc((void**)&mx, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&my, nx * ny * nz * sizeof(float));
    hipMalloc((void**)&mz, nx * ny * nz * sizeof(float));
    kernel_forwardDVF<<<gridSize, blockSize>>>(mx, my, mz, alpha_x, alpha_y, alpha_z, beta_x, beta_y, beta_z, volume, flow, nx, ny, nz);
    hipDeviceSynchronize();
    kernel_deformation2<<<gridSize, blockSize>>>(d_img1, d_img, mx, my, mz, nx, ny, nz);
    hipDeviceSynchronize();
    hipFree(mx);
    hipFree(my);
    hipFree(mz);
}

__global__ void kernel_forwardDVF(float *mx, float *my, float *mz, float *alpha_x, float *alpha_y, float *alpha_z, float *beta_x, float *beta_y, float *beta_z, float volume, float flow, int nx, int ny, int nz)
{
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    int iz = BLOCKSIZE_Z * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = ix + iy * nx + iz * nx * ny;    
    mx[id] = alpha_x[id] * volume + beta_x[id] * flow;
    my[id] = alpha_y[id] * volume + beta_y[id] * flow;
    mz[id] = alpha_z[id] * volume + beta_z[id] * flow;
}

__global__ void kernel_deformation(float *img1, hipTextureObject_t tex_img, float *mx, float *my, float *mz, int nx, int ny, int nz){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    int iz = BLOCKSIZE_Z * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = iy + ix * ny + iz * nx * ny;
    float xi = iy + 1.0f + my[id];
    float yi = ix + 1.0f + mx[id];
    float zi = iz + 1.0f + mz[id];
    img1[id] = tex3D<float>(tex_img, xi - 0.5f, yi - 0.5f, zi - 0.5f);
}

__global__ void kernel_deformation2(float *img1, float *img, float *mx, float *my, float *mz, int nx, int ny, int nz){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    int iz = BLOCKSIZE_Z * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = iy + ix * ny + iz * nx * ny;
    float dx, dy, dz;
    if (ix == nx - 1)
        dx = 0;
    else
        dx = img[id + 1] - img[id];
    if (iy == ny - 1)
        dy = 0;
    else
        dy = img[id + nx] - img[id];
    if (iz == nz - 1)
        dz = 0;
    else
        dz = img[id + nx * ny] - img[id];
    img1[id] = img[id] + dy * mx[id] + dx * my[id] + dz * mz[id];
}