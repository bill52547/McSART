#include "hip/hip_runtime.h"
#include "kernel_backprojection.h"

__host__ void kernel_backprojection(float *d_img, float *d_proj, float angle,float SO, float SD, float da, int na, float ai, float db, int nb, float bi, int nx, int ny, int nz)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    struct hipExtent extent = make_hipExtent(na, nb, 1);
    hipArray *array_proj;
    hipMalloc3DArray(&array_proj, &channelDesc, extent);
    hipMemcpy3DParms copyParams = {0};
    hipPitchedPtr dp_proj = make_hipPitchedPtr((void*) d_proj, na * sizeof(float), na, nb);
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = dp_proj;
    copyParams.dstArray = array_proj;
    hipMemcpy3D(&copyParams);

    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    resDesc.res.array.array = array_proj;
	hipTextureObject_t tex_proj = 0;
    // hipTextureObject_t tex_proj = host_create_texture_object(d_proj, nb, na, 1);
    hipCreateTextureObject(&tex_proj, &resDesc, &texDesc, NULL);

    const dim3 gridSize_img((nx + BLOCKWIDTH - 1) / BLOCKWIDTH, (ny + BLOCKHEIGHT - 1) / BLOCKHEIGHT, (nz + BLOCKDEPTH - 1) / BLOCKDEPTH);
    const dim3 blockSize(BLOCKWIDTH, BLOCKHEIGHT, BLOCKDEPTH);
	// mexPrintf("angle = %f.\n", angle);
	// mexPrintf("SO = %f.\n", SO);
	// mexPrintf("SD = %f.\n", SD);
	// mexPrintf("na = %d.\n", na);
	// mexPrintf("nb = %d.\n", nb);
	// mexPrintf("da = %f.\n", da);
	// mexPrintf("db = %f.\n", db);
	// mexPrintf("ai = %f.\n", ai);
	// mexPrintf("bi = %f.\n", bi);
    kernel<<<gridSize_img, blockSize>>>(d_img, tex_proj, angle, SO, SD, na, nb, da, db, ai, bi, nx, ny, nz);
    hipDeviceSynchronize();

    hipFreeArray(array_proj);
    hipDestroyTextureObject(tex_proj);
}


__global__ void kernel(float *img, hipTextureObject_t tex_proj, float angle, float SO, float SD, int na, int nb, float da, float db, float ai, float bi, int nx, int ny, int nz){
    int ix = BLOCKWIDTH * blockIdx.x + threadIdx.x;
    int iy = BLOCKHEIGHT * blockIdx.y + threadIdx.y;
    int iz = BLOCKDEPTH * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;

    int id = ix + iy * nx + iz * nx * ny;
	// angle += 3.141592653589793;

    img[id] = 0.0f;
	// float sphi = __sinf(angle);
	// float cphi = __cosf(angle);
	float sphi = __sinf(angle);
	float cphi = __cosf(angle);
	// float dd_voxel[3];
	float xc, yc, zc;
	xc = (float)ix - nx / 2 + 0.5f;
	yc = (float)iy - ny / 2 + 0.5f;
	zc = (float)iz - nz / 2 + 0.5f;

	// voxel boundary coordinates
	float xll, yll, zll, xlr, ylr, zlr, xrl, yrl, zrl, xrr, yrr, zrr, xt, yt, zt, xb, yb, zb;
	// xll = +(xc - 0.5f) * cphi + (yc - 0.5f) * sphi;
    // yll = -(xc - 0.5f) * sphi + (yc - 0.5f) * cphi;
    // xrr = +(xc + 0.5f) * cphi + (yc + 0.5f) * sphi;
    // yrr = -(xc + 0.5f) * sphi + (yc + 0.5f) * cphi;
    // zll = zc; zrr = zc;
	// xrl = +(xc + 0.5f) * cphi + (yc - 0.5f) * sphi;
    // yrl = -(xc + 0.5f) * sphi + (yc - 0.5f) * cphi;
    // xlr = +(xc - 0.5f) * cphi + (yc + 0.5f) * sphi;
    // ylr = -(xc - 0.5f) * sphi + (yc + 0.5f) * cphi;
    // zrl = zc; zlr = zc;
	xll = +xc * cphi + yc * sphi - 0.5f;
    yll = -xc * sphi + yc * cphi - 0.5f;
    xrr = +xc * cphi + yc * sphi + 0.5f;
    yrr = -xc * sphi + yc * cphi + 0.5f;
    zll = zc; zrr = zc;
	xrl = +xc * cphi + yc * sphi + 0.5f;
    yrl = -xc * sphi + yc * cphi - 0.5f;
    xlr = +xc * cphi + yc * sphi - 0.5f;
    ylr = -xc * sphi + yc * cphi + 0.5f;
    zrl = zc; zlr = zc;
    xt = xc * cphi + yc * sphi;
    yt = -xc * sphi + yc * cphi;
    zt = zc + 0.5f;
    xb = xc * cphi + yc * sphi;
    yb = -xc * sphi + yc * cphi;
    zb = zc - 0.5f;

	// the coordinates of source and detector plane here are after rotation
	float ratio, all, bll, alr, blr, arl, brl, arr, brr, at, bt, ab, bb, a_max, a_min, b_max, b_min;
	// calculate a value for each boundary coordinates
	

	// the a and b here are all absolute positions from isocenter, which are on detector planes
	ratio = SD / (xll + SO);
	all = ratio * yll;
	bll = ratio * zll;
	ratio = SD / (xrr + SO);
	arr = ratio * yrr;
	brr = ratio * zrr;
	ratio = SD / (xlr + SO);
	alr = ratio * ylr;
	blr = ratio * zlr;
	ratio = SD / (xrl + SO);
	arl = ratio * yrl;
	brl = ratio * zrl;
	ratio = SD / (xt + SO);
	at = ratio * yt;
	bt = ratio * zt;
	ratio = SD / (xb + SO);
	ab = ratio * yb;
	bb = ratio * zb;

	// get the max and min values of all boundary projectors of voxel boundaries on detector plane
	// a_max = MAX4(al ,ar, at, ab);
	// a_min = MIN4(al ,ar, at, ab);
	// b_max = MAX4(bl ,br, bt, bb);
	// b_min = MIN4(bl ,br, bt, bb);
	a_max = MAX6(all ,arr, alr, arl, at, ab);
	a_min = MIN6(all ,arr, alr, arl, at, ab);
	b_max = MAX6(bll ,brr, blr, brl, bt, bb);
	b_min = MIN6(bll ,brr, blr, brl, bt, bb);

	// the related positions on detector plane from start points
	a_max = a_max / da - ai + 0.5f; //  now they are the detector coordinates
	a_min = a_min / da - ai + 0.5f;
	b_max = b_max / db - bi + 0.5f;
	b_min = b_min / db - bi + 0.5f;
	int a_ind_max = (int)floorf(a_max); 	
	int a_ind_min = (int)floorf(a_min); 
	int b_ind_max = (int)floorf(b_max); 
	int b_ind_min = (int)floorf(b_min); 
	
	// int a_ind_max = (int)floorf(a_max / da - ai);
	// int a_ind_min = (int)floorf(a_min / da - ai);
	// int b_ind_max = (int)floorf(b_max / db - bi);
	// int b_ind_min = (int)floorf(b_min / db - bi);

	float bin_bound_1, bin_bound_2, wa, wb;
	for (int ia = MAX(0, a_ind_min); ia < MIN(na, a_max); ia ++){
		// bin_bound_1 = ((float)ia + ai) * da;
		// bin_bound_2 = ((float)ia + ai + 1.0f) * da;
		bin_bound_1 = ia + 0.0f;
		bin_bound_2 = ia + 1.0f;
		
		wa = MIN(bin_bound_2, a_max) - MAX(bin_bound_1, a_min);// wa /= a_max - a_min;

		for (int ib = MAX(0, b_ind_min); ib < MIN(nb, b_max); ib ++){
			// bin_bound_1 = ((float)ib + bi) * db;
			// bin_bound_2 = ((float)ib + bi + 1.0f) * db;
			bin_bound_1 = ib + 0.0f;
			bin_bound_2 = ib + 1.0f;
			// wb = MIN(bin_bound_2, b_max) - MAX(bin_bound_1, b_min);// wb /= db;
			wb = MIN(bin_bound_2, b_max) - MAX(bin_bound_1, b_min);// wb /= b_max - b_min;


			img[id] += wa * wb * tex3D<float>(tex_proj, (ia + 0.5f), (ib + 0.5f), 0.5f);
		}		
	}
}
