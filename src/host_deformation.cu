#include "host_deformation.h"
void host_deformation(float *img1, float *img, float *mx, float *my, float *mz, int nx, int ny, int nz)
{
    hipTextureObject_t tex_img = 0;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipError_t cudaStat;

    // create texture object alpha and beta
    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    struct hipExtent extent_img = make_hipExtent(nx, ny, nz);
    hipMemcpy3DParms copyParams = {0};
    copyParams.extent = extent_img;
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipArray *array_img;
    cudaStat = hipMalloc3DArray(&array_img, &channelDesc, extent_img);

    hipPitchedPtr dp_img = make_hipPitchedPtr((void*) img, nx * sizeof(float), nx, ny);
    copyParams.srcPtr = dp_img;
    copyParams.dstArray = array_img;
    cudaStat = hipMemcpy3D(&copyParams);   
    if (cudaStat != hipSuccess) {
        mexPrintf("Failed to copy dp_img to array memory array_img.\n");
        mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
            mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
    }
    resDesc.res.array.array = array_img;
    hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);
    const dim3 gridSize_img((nx + 16 - 1) / 16, (ny + 16 - 1) / 16, (nz + 4 - 1) / 4);
    const dim3 blockSize(16,16, 4);
    kernel_deformation<<<gridSize_img, blockSize>>>(img1, tex_img, mx, my, mz, nx, ny, nz);
    hipDeviceSynchronize();

    hipDestroyTextureObject(tex_img);
    hipFreeArray(array_img);
}
