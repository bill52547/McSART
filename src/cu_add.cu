#include "hip/hip_runtime.h"
#include "cu_add.h"
__host__ void host_add(float *img1, float *img, int nx, int ny, int nz, float weight){
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, (nz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
    kernel_add<<<gridSize, blockSize>>>(img1, img, nx, ny, nz, weight);
    hipDeviceSynchronize();
}

__global__ void kernel_add(float *img1, float *img, int nx, int ny, int nz, float weight){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    int iz = BLOCKSIZE_Z * blockIdx.z + threadIdx.z;
    
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = ix + iy * nx + iz * nx * ny;
    img1[id] += img[id] * weight;
}

__host__ void host_add2(float *img1, float *img, int nx, int ny, int nz, float* img0, float weight, int ind){
    const dim3 gridSize((nx + BLOCKSIZE_X - 1) / BLOCKSIZE_X, (ny + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y, (nz + BLOCKSIZE_Z - 1) / BLOCKSIZE_Z);
    const dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y, BLOCKSIZE_Z);
    kernel_add2<<<gridSize, blockSize>>>(img1, img, nx, ny, nz, img0, weight, ind);
    hipDeviceSynchronize();
    
}

__global__ void kernel_add2(float *img1, float *img, int nx, int ny, int nz, float *img0, float weight, int ind){
    int ix = BLOCKSIZE_X * blockIdx.x + threadIdx.x;
    int iy = BLOCKSIZE_Y * blockIdx.y + threadIdx.y;
    int iz = BLOCKSIZE_Z * blockIdx.z + threadIdx.z;
    
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = ix + iy * nx + iz * nx * ny;
    float df;
    switch (ind)
    {
        case 1:
            if (ix == nx - 1)
                df = 0.0f;
            else
                df = img0[id + 1] - img0[id];
            break;
        case 2:
            if (iy == ny - 1)
                df = 0.0f;
            else
                df = img0[id + nx] - img0[id];    
            break;
        case 3:
            if (iz == nz - 1)
                df = 0.0f;
            else
                df = img0[id + nx * ny] - img0[id];
            break;
    }
    img1[id] += img[iy + ix * ny + iz * nx * ny] * weight * df;
}