#include "kernel_deformation.h"
#include "mex.h"
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
// Macro for input and output
#define IN_IMG prhs[0]
#define PARA prhs[1]
#define MX prhs[2]
#define MY prhs[3]
#define MZ prhs[4]

#define OUT_IMG plhs[0]

float *h_mx, *h_my, *h_mz, *h_img;
h_img = (float*)mxGetData(IN_IMG);
h_mx = (float*)mxGetData(MX);
h_my = (float*)mxGetData(MY);
h_mz = (float*)mxGetData(MZ);
int nx, ny, nz;
nx = (int)mxGetScalar(mxGetField(PARA, 0, "nx"));
ny = (int)mxGetScalar(mxGetField(PARA, 0, "ny"));
nz = (int)mxGetScalar(mxGetField(PARA, 0, "nz"));

float *d_mx, *d_my, *d_mz, *d_img1;
hipMalloc((void**)&d_mx, nx * ny * nz * sizeof(float));
hipMalloc((void**)&d_my, nx * ny * nz * sizeof(float));
hipMalloc((void**)&d_mz, nx * ny * nz * sizeof(float));
hipMalloc((void**)&d_img1, nx * ny * nz * sizeof(float));

hipMemcpy(d_mx, h_mx, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_my, h_my, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_mz, h_mz, nx * ny * nz * sizeof(float), hipMemcpyHostToDevice);

OUT_IMG = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS, mxREAL);
mwSize outDim[3] = {(mwSize)nx, (mwSize)ny, (mwSize)nz};
mxSetDimensions(OUT_IMG, outDim, 3);
mxSetData(OUT_IMG, mxMalloc(nx * ny * nz * sizeof(float)));
float *h_outimg = (float*)mxGetData(OUT_IMG);


hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipPitchedPtr dp_img = make_hipPitchedPtr((void*) h_img, nx * sizeof(float), nx, ny);
hipMemcpy3DParms copyParams = {0};
struct hipExtent extent_img = make_hipExtent(nx, ny, nz);
copyParams.extent = extent_img;
copyParams.kind = hipMemcpyHostToDevice;
copyParams.srcPtr = dp_img;
hipArray *array_img;
hipMalloc3DArray(&array_img, &channelDesc, extent_img);
copyParams.dstArray = array_img;
hipMemcpy3D(&copyParams);   

hipResourceDesc resDesc;
hipTextureDesc texDesc;
memset(&resDesc, 0, sizeof(resDesc));
resDesc.resType = hipResourceTypeArray;
memset(&texDesc, 0, sizeof(texDesc));
texDesc.addressMode[0] = hipAddressModeClamp;
texDesc.addressMode[1] = hipAddressModeClamp;
texDesc.addressMode[2] = hipAddressModeClamp;
texDesc.filterMode = hipFilterModeLinear;
texDesc.readMode = hipReadModeElementType;
texDesc.normalizedCoords = 0;
resDesc.res.array.array = array_img;
hipTextureObject_t tex_img = 0;
hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);


const dim3 gridSize_img((nx + 16 - 1) / 16, (ny + 16 - 1) / 16, (nz + 4 - 1) / 4);
const dim3 blockSize(16, 16, 4);
kernel_deformation<<<gridSize_img, blockSize>>>(d_img1, tex_img, d_mx, d_my, d_mz, nx, ny, nz);
hipDeviceSynchronize();

hipMemcpy(h_outimg, d_img1, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost);

hipFreeArray(array_img);
hipFree(d_mx);
hipFree(d_my);
hipFree(d_mz);
hipFree(d_img1);

hipDeviceReset();
return;
}

