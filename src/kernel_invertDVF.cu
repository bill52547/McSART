
#include <hip/hip_runtime.h>
__global__ void kernel_invertDVF(float *mx2, float *my2, float *mz2, hipTextureObject_t mx, hipTextureObject_t my, hipTextureObject_t mz, int nx, int ny, int nz, int niter);
__host__ void host_invertDVF(float *mx2, float *my2, float *mz2, float *mx, float *my, float *mz, int nx, int ny, int nz, int niter)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    struct hipExtent extent_img = make_hipExtent(nx, ny, nz);

    hipPitchedPtr dp_mx = make_hipPitchedPtr((void*) mx, nx * sizeof(float), nx, ny);
    hipPitchedPtr dp_my = make_hipPitchedPtr((void*) my, nx * sizeof(float), nx, ny);
    hipPitchedPtr dp_mz = make_hipPitchedPtr((void*) mz, nx * sizeof(float), nx, ny);
    hipMemcpy3DParms copyParams = {0};
    copyParams.extent = extent_img;
    copyParams.kind = hipMemcpyHostToDevice;
    hipArray *array_mx, *array_my, *array_mz;

    hipMalloc3DArray(&array_mx, &channelDesc, extent_img);
    hipMalloc3DArray(&array_my, &channelDesc, extent_img);
    hipMalloc3DArray(&array_mz, &channelDesc, extent_img);

    copyParams.srcPtr = dp_mx;
    copyParams.dstArray = array_mx;
    hipMemcpy3D(&copyParams);  

    copyParams.srcPtr = dp_my;
    copyParams.dstArray = array_my;
    hipMemcpy3D(&copyParams);  

    copyParams.srcPtr = dp_mz;
    copyParams.dstArray = array_mz;
    hipMemcpy3D(&copyParams);   

    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
        resDesc.res.array.array = array_mx;
    hipTextureObject_t tex_mx = 0;
    hipCreateTextureObject(&tex_mx, &resDesc, &texDesc, NULL);
        resDesc.res.array.array = array_my;
    hipTextureObject_t tex_my = 0;
    hipCreateTextureObject(&tex_my, &resDesc, &texDesc, NULL);
        resDesc.res.array.array = array_mz;
    hipTextureObject_t tex_mz = 0;
    hipCreateTextureObject(&tex_mz, &resDesc, &texDesc, NULL);

    const dim3 gridSize_img((nx + 16 - 1) / 16, (ny + 16 - 1) / 16, (nz + 4 - 1) / 4);
    const dim3 blockSize(16, 16, 4);
    kernel_invertDVF<<<gridSize_img, blockSize>>>(mx2, my2, mz2, tex_mx, tex_my, tex_mz, nx, ny, nz, niter);
hipDeviceSynchronize();


hipDestroyTextureObject(tex_mx);
hipDestroyTextureObject(tex_my);
hipDestroyTextureObject(tex_mz);

hipFreeArray(array_mx);
hipFreeArray(array_my);
hipFreeArray(array_mz);

return;
}
__global__ void kernel_invertDVF(float *mx2, float *my2, float *mz2, hipTextureObject_t mx, hipTextureObject_t my, hipTextureObject_t mz, int nx, int ny, int nz, int niter)
{
    int ix = 16 * blockIdx.x + threadIdx.x;
    int iy = 16 * blockIdx.y + threadIdx.y;
    int iz = 4 * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = ix + iy * nx + iz * nx * ny;
    float x = 0, y = 0, z = 0;
    for (int iter = 0; iter < niter; iter ++){
        x = - tex3D<float>(mx, (x + ix + 0.5f), (y + iy + 0.5f), (z + iz + 0.5f));
        y = - tex3D<float>(my, (x + ix + 0.5f), (y + iy + 0.5f), (z + iz + 0.5f));
        z = - tex3D<float>(mz, (x + ix + 0.5f), (y + iy + 0.5f), (z + iz + 0.5f));
    }
    mx2[id] = x;
    my2[id] = y;
    mz2[id] = z;
}