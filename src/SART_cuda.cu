#include "SART_cuda.h" // consists all required package and functions

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
// Macro for input and output
#define IN_IMG prhs[0]
#define PROJ prhs[1]
#define GEO_PARA prhs[2]
#define ITER_PARA prhs[3]
#define OUT_IMG plhs[0]
// #define OUT_ERR plhs[1]

int nx, ny, nz, na, nb, numImg, numBytesImg, numSingleProj, numBytesSingleProj;
float da, db, ai, bi, SO, SD, dx;

// resolutions of volumes 
if (mxGetField(GEO_PARA, 0, "nx") != NULL)
    nx = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nx"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution nx.\n");

if (mxGetField(GEO_PARA, 0, "ny") != NULL)
    ny = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "ny"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution ny.\n");

if (mxGetField(GEO_PARA, 0, "nz") != NULL)
    nz = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nz"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution nz.\n");

numImg = nx * ny * nz; // size of image
numBytesImg = numImg * sizeof(float); // number of bytes in image

// detector plane resolutions
if (mxGetField(GEO_PARA, 0, "na") != NULL)
    na = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "na"));
else if (mxGetField(GEO_PARA, 0, "nv") != NULL)
    na = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nv"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number of detector in plane, which is denoted as na or nu.\n");

if (mxGetField(GEO_PARA, 0, "nb") != NULL)
    nb = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nb"));
else if (mxGetField(GEO_PARA, 0, "nu") != NULL)
    nb = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nu"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number of detector across plane, which is denoted as nb or nv.\n");

numSingleProj = na * nb;
numBytesSingleProj = numSingleProj * sizeof(float);

// voxel resolution dx, which is also the scaling factor of the whole system
if (mxGetField(GEO_PARA, 0, "dx") != NULL)
    dx = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "dx"));
else{
    dx = 1.0f;
    mexPrintf("Automatically set voxel size dx to 1. \n");
    mexPrintf("If don't want that default value, please set para.dx manually.\n");
}

// detector resolution
if (mxGetField(GEO_PARA, 0, "da") != NULL)
    da = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "da"));
else{
    da = 1.0f;
    mexPrintf("Automatically set detector cell size da to 1. \n");
    mexPrintf("If don't want that default value, please set para.da manually.\n");
}

if (mxGetField(GEO_PARA, 0, "db") != NULL)
    db = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "db"));
else{
    db = 1.0f;
    mexPrintf("Automatically set detectof cell size db to 1. \n");
    mexPrintf("If don't want that default value, please set para.db manually.\n");
}


// detector plane offset from centered calibrations
if (mxGetField(GEO_PARA, 0, "ai") != NULL){
    ai = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "ai"));
    ai -= ((float)na / 2 - 0.5f);
}
else{
    mexPrintf("Automatically set detector offset ai to 0. \n");
    mexPrintf("If don't want that default value, please set para.ai manually.\n");
    ai = - ((float)na / 2 - 0.5f);
}

if (mxGetField(GEO_PARA, 0, "bi") != NULL){
    bi = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "bi"));
    bi -= ((float)nb / 2 - 0.5f);
}
else{
    mexPrintf("Automatically set detector offset bi to 0. \n");
    mexPrintf("If don't want that default value, please set para.bi manually.\n");
    bi = - ((float)nb / 2 - 0.5f);
}


if (mxGetField(GEO_PARA, 0, "SO") != NULL)
    SO = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SO"));
else if (mxGetField(GEO_PARA, 0, "SI") != NULL)
    SO = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SI"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid distance between source and isocenter, which is denoted with para.SO or para.DI.\n");

if (mxGetField(GEO_PARA, 0, "SD") != NULL)
    SD = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SD"));
else if (mxGetField(GEO_PARA, 0, "DI") != NULL)
    SD = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "DI")) + SO;
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid distance between source and detector plane, which is denoted with para.SD or para.SI + para.DI.\n");


// load iterating parameters, for the whole bin
int n_iter, n_iter_invertDVF;
if (mxGetField(ITER_PARA, 0, "n_iter") != NULL)
    n_iter = (int)mxGetScalar(mxGetField(ITER_PARA, 0, "n_iter")); // number of views in this bin
else{
    n_iter = 1;
    mexPrintf("Automatically set number of iterations to 1. \n");
    mexPrintf("If don't want that default value, please set iter_para.n_iter manually.\n");
}

if (mxGetField(ITER_PARA, 0, "n_iter_invertDVF") != NULL)
    n_iter_invertDVF = (int)mxGetScalar(mxGetField(ITER_PARA, 0, "n_iter_invertDVF"));
else{
    n_iter_invertDVF = 10;
    mexPrintf("Automatically set number of iterations for inverting DVF to 10. \n");
    mexPrintf("If don't want that default value, please set iter_para.n_iter_invertDVF manually.\n");
}

int n_bin, *n_views, numProj, numBytesProj, N_view; // number of bins, numbers of views of bins, and the index view of each bin.
// e.g. we have 3 bins here with 10 total views. For these 3 bins, they holds 1,3,6 views. Then we will set n_views as {0, 1, 4, 10}, which is the starting view indices of each bin. Moreover, we need to well arrange the volumes and flows.

if (mxGetField(ITER_PARA, 0, "n_bin") != NULL)
    n_bin = (int)mxGetScalar(mxGetField(ITER_PARA, 0, "n_bin"));
else{
    n_bin = 8;
    mexPrintf("Automatically set number of bins to 8. \n");
    mexPrintf("If don't want that default value, please set iter_para.n_bin manually.\n");
}

if (mxGetField(ITER_PARA, 0, "n_views") != NULL)
    n_views = (int*)mxGetData(mxGetField(ITER_PARA, 0, "n_views"));
else{
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number bins, which is denoted as iter_para.n_views.\n");
}
N_view = n_views[n_bin];

// 5D models
float *h_alpha_x, *h_alpha_y, *h_alpha_z, *h_beta_x, *h_beta_y, *h_beta_z, *angles, lambda;

// load 5DCT alpha and beta
if (mxGetField(ITER_PARA, 0, "alpha_x") != NULL)
    h_alpha_x = (float*)mxGetData(mxGetField(ITER_PARA, 0, "alpha_x")); 
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.alpha_x.\n");    

if (mxGetField(ITER_PARA, 0, "alpha_y") != NULL)
    h_alpha_y = (float*)mxGetData(mxGetField(ITER_PARA, 0, "alpha_y")); 
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.alpha_y.\n");

if (mxGetField(ITER_PARA, 0, "alpha_z") != NULL)
    h_alpha_z = (float*)mxGetData(mxGetField(ITER_PARA, 0, "alpha_z"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.alpha_z.\n");

if (mxGetField(ITER_PARA, 0, "beta_x") != NULL)
    h_beta_x = (float*)mxGetData(mxGetField(ITER_PARA, 0, "beta_x"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.beta_x.\n");

if (mxGetField(ITER_PARA, 0, "beta_y") != NULL)
    h_beta_y = (float*)mxGetData(mxGetField(ITER_PARA, 0, "beta_y")); 
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.beta_y.\n");

if (mxGetField(ITER_PARA, 0, "beta_z") != NULL)
    h_beta_z = (float*)mxGetData(mxGetField(ITER_PARA, 0, "beta_z"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid iter_para.beta_z.\n");

// load 5DCT parameters volume (v) and flow (f)
float *volumes, *flows, *ref_volumes, *ref_flows;
if (mxGetField(ITER_PARA, 0, "volumes") != NULL)
    volumes= (float*)mxGetData(mxGetField(ITER_PARA, 0, "volumes"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume in iter_para.volumes.\n");  

if (mxGetField(ITER_PARA, 0, "flows") != NULL)
    flows = (float*)mxGetData(mxGetField(ITER_PARA, 0, "flows"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid flow in iter_para.flows.\n");    

if (mxGetField(ITER_PARA, 0, "volume0") != NULL)
    ref_volumes = (float*)mxGetData(mxGetField(ITER_PARA, 0, "volume0"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid referenced volume in iter_para.volume0.\n");    

if (mxGetField(ITER_PARA, 0, "flow0") != NULL)
    ref_flows = (float*)mxGetData(mxGetField(ITER_PARA, 0, "flow0"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid referenced flow in iter_para.flow0.\n");    

if (mxGetField(ITER_PARA, 0, "angles") != NULL)
    angles = (float*)mxGetData(mxGetField(ITER_PARA, 0, "angles"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid angles iter_para.angles.\n");
if (mxGetField(ITER_PARA, 0, "lambda") != NULL)
    lambda = (float)mxGetScalar(mxGetField(ITER_PARA, 0, "lambda"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid coefficience iter_para.lambda.\n");

numProj = numSingleProj * N_view;
numBytesProj = numProj * sizeof(float);

// load initial guess of image
float *h_img;
h_img = (float*)mxGetData(IN_IMG);

// load true projection value
float *h_proj;
h_proj = (float*)mxGetData(PROJ);

// define thread distributions
const dim3 gridSize_img((nx + BLOCKWIDTH - 1) / BLOCKWIDTH, (ny + BLOCKHEIGHT - 1) / BLOCKHEIGHT, (nz + BLOCKDEPTH - 1) / BLOCKDEPTH);
const dim3 gridSize_singleProj((na + BLOCKWIDTH - 1) / BLOCKWIDTH, (nb + BLOCKHEIGHT - 1) / BLOCKHEIGHT, 1);
const dim3 blockSize(BLOCKWIDTH,BLOCKHEIGHT, BLOCKDEPTH);

// CUDA 3DArray Malloc parameters
struct hipExtent extent_img = make_hipExtent(nx, ny, nz);
struct hipExtent extent_singleProj = make_hipExtent(na, nb, 1);

//Allocate CUDA array in device memory of 5DCT matrices: alpha and beta
hipArray *d_alpha_x, *d_alpha_y, *d_alpha_z, *d_beta_x, *d_beta_y, *d_beta_z;
hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

hipError_t cudaStat;
// alpha_x
cudaStat = hipMalloc3DArray(&d_alpha_x, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for alpha_x failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

// alpha_y
cudaStat = hipMalloc3DArray(&d_alpha_y, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for alpha_y failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

// alpha_z
cudaStat = hipMalloc3DArray(&d_alpha_z, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for alpha_z failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

// beta_x
cudaStat = hipMalloc3DArray(&d_beta_x, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for beta_x failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}
// beta_y
cudaStat = hipMalloc3DArray(&d_beta_y, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for beta_y failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}
// beta_z
cudaStat = hipMalloc3DArray(&d_beta_z, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for beta_z failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}


// Get pitched pointer to alpha and beta in host memory
hipPitchedPtr hp_alpha_x = make_hipPitchedPtr((void*) h_alpha_x, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_alpha_y = make_hipPitchedPtr((void*) h_alpha_y, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_alpha_z = make_hipPitchedPtr((void*) h_alpha_z, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_beta_x = make_hipPitchedPtr((void*) h_beta_x, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_beta_y = make_hipPitchedPtr((void*) h_beta_y, nx * sizeof(float), nx, ny);
hipPitchedPtr hp_beta_z = make_hipPitchedPtr((void*) h_beta_z, nx * sizeof(float), nx, ny);

// Copy alpha and beta to texture memory from pitched pointer
hipMemcpy3DParms copyParams = {0};
copyParams.extent = extent_img;
copyParams.kind = hipMemcpyHostToDevice;

//alpha_x
copyParams.srcPtr = hp_alpha_x;
copyParams.dstArray = d_alpha_x;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy alpha_x to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//alpha_y
copyParams.srcPtr = hp_alpha_y;
copyParams.dstArray = d_alpha_y;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy alpha_y to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//alpha_z
copyParams.srcPtr = hp_alpha_z;
copyParams.dstArray = d_alpha_z;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy alpha_z to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//beta_x
copyParams.srcPtr = hp_beta_x;
copyParams.dstArray = d_beta_x;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy beta_x to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//beta_y
copyParams.srcPtr = hp_beta_y;
copyParams.dstArray = d_beta_y;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy beta_y to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

//beta_z
copyParams.srcPtr = hp_beta_z;
copyParams.dstArray = d_beta_z;
cudaStat = hipMemcpy3D(&copyParams);
if (cudaStat != hipSuccess) {
	mexPrintf("Failed to copy beta_z to device memory.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}


// create texture object alpha and beta
hipResourceDesc resDesc;
hipTextureDesc texDesc, texDesc2;
memset(&resDesc, 0, sizeof(resDesc));
resDesc.resType = hipResourceTypeArray;

memset(&texDesc, 0, sizeof(texDesc));
texDesc.addressMode[0] = hipAddressModeClamp;
texDesc.addressMode[1] = hipAddressModeClamp;
texDesc.addressMode[2] = hipAddressModeClamp;
texDesc.filterMode = hipFilterModeLinear;
texDesc.readMode = hipReadModeElementType;
texDesc.normalizedCoords = 0;

memset(&texDesc2, 0, sizeof(texDesc2));
texDesc2.addressMode[0] = hipAddressModeClamp;
texDesc2.addressMode[1] = hipAddressModeClamp;
texDesc2.addressMode[2] = hipAddressModeClamp;
texDesc2.filterMode = hipFilterModePoint;
texDesc2.readMode = hipReadModeElementType;
texDesc2.normalizedCoords = 0;

// alpha_x
resDesc.res.array.array = d_alpha_x;
hipTextureObject_t tex_alpha_x = 0;
hipCreateTextureObject(&tex_alpha_x, &resDesc, &texDesc, NULL);

// alpha_y
resDesc.res.array.array = d_alpha_y;
hipTextureObject_t tex_alpha_y = 0;
hipCreateTextureObject(&tex_alpha_y, &resDesc, &texDesc, NULL);

// alpha_z
resDesc.res.array.array = d_alpha_z;
hipTextureObject_t tex_alpha_z = 0;
hipCreateTextureObject(&tex_alpha_z, &resDesc, &texDesc, NULL);

// beta_x
resDesc.res.array.array = d_beta_x;
hipTextureObject_t tex_beta_x = 0;
hipCreateTextureObject(&tex_beta_x, &resDesc, &texDesc, NULL);

// beta_y
resDesc.res.array.array = d_beta_y;
hipTextureObject_t tex_beta_y = 0;
hipCreateTextureObject(&tex_beta_y, &resDesc, &texDesc, NULL);

// beta_z
resDesc.res.array.array = d_beta_z;
hipTextureObject_t tex_beta_z = 0;
hipCreateTextureObject(&tex_beta_z, &resDesc, &texDesc, NULL);

// malloc in device: projection of the whole bin
float *d_proj;
hipMalloc((void**)&d_proj, numBytesSingleProj);

// malloc in device: another projection pointer, with single view size
float *d_singleViewProj2;
hipMalloc((void**)&d_singleViewProj2, numBytesSingleProj);

// malloc in device: projection of the whole bin
float *d_img ,*d_img1;
hipArray *array_img;
hipMalloc((void**)&d_img, numBytesImg);
hipMalloc((void**)&d_img1, numBytesImg);
cudaStat = hipMalloc3DArray(&array_img, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_img failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

// malloc in device: another image pointer, for single view 
float *d_singleViewImg, *d_imgOnes;
hipMalloc(&d_singleViewImg, numBytesImg);
hipMalloc(&d_imgOnes, numBytesImg);
float angle, volume, flow;

//Malloc forward and inverted DVFs in device
float *d_mx, *d_my, *d_mz;
hipMalloc(&d_mx, numBytesImg);
hipMalloc(&d_my, numBytesImg);
hipMalloc(&d_mz, numBytesImg);


// Alloc forward and inverted DVFs in device, in form of array memory
hipArray *array_mx, *array_my, *array_mz;
cudaStat = hipMalloc3DArray(&array_mx, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_mx failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

cudaStat = hipMalloc3DArray(&array_my, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_my failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

cudaStat = hipMalloc3DArray(&array_mz, &channelDesc, extent_img);
if (cudaStat != hipSuccess) {
	mexPrintf("Array memory allocation for array_mz failed.\n");
	mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
        mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
}

// define tex_mx etc
hipTextureObject_t tex_mx = 0, tex_my = 0, tex_mz = 0;

// setup output images
OUT_IMG = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS, mxREAL);
const mwSize outDim[4] = {(mwSize)nx, (mwSize)ny, (mwSize)nz, (mwSize)n_bin};
mxSetDimensions(OUT_IMG, outDim, 4);
mxSetData(OUT_IMG, mxMalloc(numBytesImg * n_bin));
float *h_outimg = (float*)mxGetData(OUT_IMG);


copyParams.kind = hipMemcpyDeviceToDevice;

for (int ibin = 0; ibin < n_bin; ibin++){
    if (ibin < 1){
        hipMemcpy(d_img, h_img, numBytesImg, hipMemcpyHostToDevice);
    }
    else{
    //     // hipMemcpy(d_img1, h_img, numBytesImg, hipMemcpyHostToDevice);
        volume = ref_volumes[ibin] - ref_volumes[0];
        flow = ref_flows[ibin] - ref_flows[0];
        kernel_forwardDVF<<<gridSize_img, blockSize>>>(d_mx, d_my, d_mz, tex_alpha_x, tex_alpha_y, tex_alpha_z, tex_beta_x, tex_beta_y, tex_beta_z, volume, flow, nx, ny, nz);
        hipDeviceSynchronize();

        // copy img to pitched pointer and bind it to a texture object
        hipPitchedPtr dp_img = make_hipPitchedPtr((void*) d_img1, nx * sizeof(float), nx, ny);
        copyParams.srcPtr = dp_img;
        copyParams.dstArray = array_img;
        cudaStat = hipMemcpy3D(&copyParams);   
        if (cudaStat != hipSuccess) {
            mexPrintf("Failed to copy dp_img to array memory array_img.\n");
            mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
        }
        resDesc.res.array.array = array_img;
        hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);

        kernel_deformation<<<gridSize_img, blockSize>>>(d_img, tex_img, d_mx, d_my, d_mz, nx, ny, nz);
        hipDeviceSynchronize();
    }
    for (int iter = 0; iter < n_iter; iter++){ // iteration
        processBar(ibin, n_bin, iter, n_iter);
        
        for (int i_view = n_views[ibin]; i_view < n_views[ibin + 1]; i_view++){ // view
        
            angle = angles[i_view];
            volume = ref_volumes[ibin] - ref_volumes[0]
            flow = ref_flows[ibin] - ref_flows[0];
            
            kernel_forwardDVF<<<gridSize_img, blockSize>>>(d_mx, d_my, d_mz, tex_alpha_x, tex_alpha_y, tex_alpha_z, tex_beta_x, tex_beta_y, tex_beta_z, volume, flow, nx, ny, nz);
            hipDeviceSynchronize();
            
            // copy mx etc to pitched pointer and bind it to a texture object
            hipPitchedPtr dp_mx = make_hipPitchedPtr((void*) d_mx, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_mx;
            copyParams.dstArray = array_mx;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_mx to array memory array_mx.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_mx;
            hipCreateTextureObject(&tex_mx, &resDesc, &texDesc, NULL);

            hipPitchedPtr dp_my = make_hipPitchedPtr((void*) d_my, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_my;
            copyParams.dstArray = array_my;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_my to array memory array_my.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_my;
            hipCreateTextureObject(&tex_my, &resDesc, &texDesc, NULL);

            hipPitchedPtr dp_mz = make_hipPitchedPtr((void*) d_mz, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_mz;
            copyParams.dstArray = array_mz;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_mz to array memory array_mz.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_mz;
            hipCreateTextureObject(&tex_mz, &resDesc, &texDesc, NULL);

            kernel_invertDVF<<<gridSize_img, blockSize>>>(d_mx, d_my, d_mz, tex_mx, tex_my, tex_mz, nx, ny, nz, n_iter_invertDVF);
            hipDeviceSynchronize();        
            
            // copy img to pitched pointer and bind it to a texture object
            hipPitchedPtr dp_img = make_hipPitchedPtr((void*) d_img, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_img;
            copyParams.dstArray = array_img;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_img to array memory array_img.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_img;
            hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);

            kernel_deformation<<<gridSize_img, blockSize>>>(d_singleViewImg, tex_img, d_mx, d_my, d_mz, nx, ny, nz); // d_singleViewImg is for ref-phase 0
            hipDeviceSynchronize();

            volume = volumes[i_view] - ref_volumes[0]
            flow = flows[i_view] - ref_flows[0];
            
            // generate forwards DVFs: d_mx, d_my, d_mz and inverted DVFs: d_mx, d_my, d_mz
            kernel_forwardDVF<<<gridSize_img, blockSize>>>(d_mx, d_my, d_mz, tex_alpha_x, tex_alpha_y, tex_alpha_z, tex_beta_x, tex_beta_y, tex_beta_z, volume, flow, nx, ny, nz);
            hipDeviceSynchronize();

            // copy img to pitched pointer and bind it to a texture object
            hipPitchedPtr dp_img = make_hipPitchedPtr((void*) d_singleViewImg, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_img;
            copyParams.dstArray = array_img;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_img to array memory array_img.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_img;
            hipCreateTextureObject(&tex_img, &resDesc, &texDesc, NULL);

            kernel_deformation<<<gridSize_img, blockSize>>>(d_singleViewImg, tex_img, d_mx, d_my, d_mz, nx, ny, nz); // d_singleViewImg is for ref-phase 0
            hipDeviceSynchronize();

            // projection of deformed image from initial guess
            kernel_projection<<<gridSize_singleProj, blockSize>>>(d_singleViewProj2, d_singleViewImg, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz); // TBD
            hipDeviceSynchronize();

            // difference between true projection and projection from initial guess
            // update d_singleViewProj2 instead of malloc a new one
            hipMemcpy(d_proj, h_proj + i_view * numSingleProj, numBytesSingleProj, hipMemcpyHostToDevice);

            kernel_add<<<gridSize_singleProj, blockSize>>>(d_singleViewProj2, d_proj, 0, na, nb, -1);
            hipDeviceSynchronize();

            // backprojecting the difference of projections
            kernel_backprojection(d_singleViewImg, d_singleViewProj2, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);

            // calculate the ones backprojection data
            kernel_initial<<<gridSize_img, blockSize>>>(d_imgOnes, nx, ny, nz, 1);
            hipDeviceSynchronize();

            kernel_projection<<<gridSize_singleProj, blockSize>>>(d_singleViewProj2, d_imgOnes, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
            hipDeviceSynchronize();

            kernel_backprojection(d_imgOnes, d_singleViewProj2, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);

            // weighting
            kernel_division<<<gridSize_img, blockSize>>>(d_singleViewImg, d_imgOnes, nx, ny, nz);
            hipDeviceSynchronize();
            
            // copy mx etc to pitched pointer and bind it to a texture object
            hipPitchedPtr dp_mx = make_hipPitchedPtr((void*) d_mx, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_mx;
            copyParams.dstArray = array_mx;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_mx to array memory array_mx.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_mx;
            hipCreateTextureObject(&tex_mx, &resDesc, &texDesc, NULL);

            hipPitchedPtr dp_my = make_hipPitchedPtr((void*) d_my, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_my;
            copyParams.dstArray = array_my;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_my to array memory array_my.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_my;
            hipCreateTextureObject(&tex_my, &resDesc, &texDesc, NULL);

            hipPitchedPtr dp_mz = make_hipPitchedPtr((void*) d_mz, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_mz;
            copyParams.dstArray = array_mz;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_mz to array memory array_mz.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_mz;
            hipCreateTextureObject(&tex_mz, &resDesc, &texDesc, NULL);
            
            kernel_invertDVF<<<gridSize_img, blockSize>>>(d_mx, d_my, d_mz, tex_mx, tex_my, tex_mz, nx, ny, nz, n_iter_invertDVF);
            hipDeviceSynchronize();   
            
            // copy img to pitched pointer and bind it to a texture object
            dp_img = make_hipPitchedPtr((void*) d_singleViewImg, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_img;
            copyParams.dstArray = array_img;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_img to array memory array_img.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_img;
            hipCreateTextureObject(&tex_img, &resDesc, &texDesc2, NULL);

            kernel_deformation<<<gridSize_img, blockSize>>>(d_singleViewImg, tex_img, d_mx, d_my, d_mz, nx, ny, nz);
            hipDeviceSynchronize();

            // copy img to pitched pointer and bind it to a texture object
            dp_img = make_hipPitchedPtr((void*) d_singleViewImg, nx * sizeof(float), nx, ny);
            copyParams.srcPtr = dp_img;
            copyParams.dstArray = array_img;
            cudaStat = hipMemcpy3D(&copyParams);   
            if (cudaStat != hipSuccess) {
                mexPrintf("Failed to copy dp_img to array memory array_img.\n");
                mexPrintf("Error code %d: %s\n",cudaStat,hipGetErrorString(cudaStat));
                    mexErrMsgIdAndTxt("MATLAB:cudaFail","SART failed.\n");
            }
            resDesc.res.array.array = array_img;
            hipCreateTextureObject(&tex_img, &resDesc, &texDesc2, NULL);
            
            volume = ref_volumes[ibin] - ref_volumes[0]
            flow = ref_flows[ibin] - ref_flows[0];
            
            kernel_forwardDVF<<<gridSize_img, blockSize>>>(d_mx, d_my, d_mz, tex_alpha_x, tex_alpha_y, tex_alpha_z, tex_beta_x, tex_beta_y, tex_beta_z, volume, flow, nx, ny, nz);
            hipDeviceSynchronize();

            kernel_deformation<<<gridSize_img, blockSize>>>(d_singleViewImg, tex_img, d_mx, d_my, d_mz, nx, ny, nz);
            hipDeviceSynchronize();

            // updating
            kernel_update<<<gridSize_img, blockSize>>>(d_img, d_singleViewImg, nx, ny, nz, lambda);
            hipDeviceSynchronize();          
        }  
    }
    if (ibin == 0){
        hipMemcpy(d_img1, d_img, numBytesImg, hipMemcpyDeviceToDevice);
    } 
    hipMemcpy(h_outimg + ibin * numImg, d_img, numBytesImg, hipMemcpyDeviceToHost);    
}


hipDestroyTextureObject(tex_alpha_x);
hipDestroyTextureObject(tex_alpha_y);
hipDestroyTextureObject(tex_alpha_z);
hipDestroyTextureObject(tex_beta_x);
hipDestroyTextureObject(tex_beta_y);
hipDestroyTextureObject(tex_beta_z);
hipDestroyTextureObject(tex_img);
hipDestroyTextureObject(tex_mx);
hipDestroyTextureObject(tex_my);
hipDestroyTextureObject(tex_mz);

hipFreeArray(d_alpha_x);
hipFreeArray(d_alpha_y);
hipFreeArray(d_alpha_z);
hipFreeArray(d_beta_x);
hipFreeArray(d_beta_y);
hipFreeArray(d_beta_z);
// hipFreeArray(d_img);
hipFree(d_mx);
hipFree(d_my);
hipFree(d_mz);
hipFreeArray(array_mx);
hipFreeArray(array_my);
hipFreeArray(array_mz);
hipFree(d_proj);
hipFree(d_singleViewImg);
hipFree(d_singleViewProj2);

hipFree(d_img);
hipFree(d_img1);
hipDeviceReset();
return;
}

