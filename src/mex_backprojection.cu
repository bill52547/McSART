#include "cu_backprojection.h" // consists all required package and functions

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
// Macro for input and output
#define IN_PROJ prhs[0]
#define GEO_PARA prhs[1]
#define OUT_IMG plhs[0]

int nx, ny, nz, na, nb, numImg, numBytesImg, numSingleProj, numBytesSingleProj;
float da, db, ai, bi, SO, SD, angle;

// resolutions of volumes 
if (mxGetField(GEO_PARA, 0, "nx") != NULL)
    nx = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nx"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution nx.\n");

if (mxGetField(GEO_PARA, 0, "ny") != NULL)
    ny = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "ny"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution ny.\n");

if (mxGetField(GEO_PARA, 0, "nz") != NULL)
    nz = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nz"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution nz.\n");

numImg = nx * ny * nz; // size of image
numBytesImg = numImg * sizeof(float); // number of bytes in image

// detector plane resolutions
if (mxGetField(GEO_PARA, 0, "na") != NULL)
    na = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "na"));
else if (mxGetField(GEO_PARA, 0, "nv") != NULL)
    na = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nv"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number of detector in plane, which is denoted as na or nu.\n");

if (mxGetField(GEO_PARA, 0, "nb") != NULL)
    nb = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nb"));
else if (mxGetField(GEO_PARA, 0, "nu") != NULL)
    nb = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nu"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number of detector across plane, which is denoted as nb or nv.\n");

numSingleProj = na * nb;
numBytesSingleProj = numSingleProj * sizeof(float);

// detector resolution
if (mxGetField(GEO_PARA, 0, "da") != NULL)
    da = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "da"));
else{
    da = 1.0f;
    mexPrintf("Automatically set detector cell size da to 1. \n");
    mexPrintf("If don't want that default value, please set para.da manually.\n");
}

if (mxGetField(GEO_PARA, 0, "db") != NULL)
    db = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "db"));
else{
    db = 1.0f;
    mexPrintf("Automatically set detectof cell size db to 1. \n");
    mexPrintf("If don't want that default value, please set para.db manually.\n");
}


// detector plane offset from centered calibrations
if (mxGetField(GEO_PARA, 0, "ai") != NULL){
    ai = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "ai"));
    ai -= ((float)na / 2 - 0.5f);
}
else{
    mexPrintf("Automatically set detector offset ai to 0. \n");
    mexPrintf("If don't want that default value, please set para.ai manually.\n");
    ai = - (float)na / 2 + 0.5f;
}

if (mxGetField(GEO_PARA, 0, "bi") != NULL){
    bi = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "bi"));
    // if (bi > -1)
    bi -= ((float)nb / 2 - 0.5f);
}
else{
    mexPrintf("Automatically set detector offset bi to 0. \n");
    mexPrintf("If don't want that default value, please set para.bi manually.\n");
    bi = - (float)nb / 2 + 0.5f;
}


if (mxGetField(GEO_PARA, 0, "SO") != NULL)
    SO = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SO"));
else if (mxGetField(GEO_PARA, 0, "SI") != NULL)
    SO = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SI"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid distance between source and isocenter, which is denoted with para.SO or para.DI.\n");

if (mxGetField(GEO_PARA, 0, "SD") != NULL)
    SD = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SD"));
else if (mxGetField(GEO_PARA, 0, "DI") != NULL)
    SD = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "DI")) + SO;
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid distance between source and detector plane, which is denoted with para.SD or para.SI + para.DI.\n");

if (mxGetField(GEO_PARA, 0, "angle") != NULL)
    angle = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "angle"));
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid projection angle, which is denoted with para.angle.\n");

float *d_img, *d_proj;
hipMalloc((void**)&d_img, nx * ny * nz * sizeof(float));
hipMalloc((void**)&d_proj, na * nb * sizeof(float));

float *h_proj;
h_proj = (float*)mxGetData(IN_PROJ);

hipMemcpy(d_proj, h_proj, na * nb * sizeof(float), hipMemcpyHostToDevice);



host_backprojection(d_img, d_proj, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);

OUT_IMG = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS, mxREAL);
const mwSize outDim[3] = {(mwSize)nx, (mwSize)ny, (mwSize)nz};

mxSetDimensions(OUT_IMG, outDim, 3);
mxSetData(OUT_IMG, mxMalloc(nx * ny * nz * sizeof(float)));
float *h_outimg = (float*)mxGetData(OUT_IMG);

hipMemcpy(h_outimg, d_img, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost);

hipFree(d_proj);
hipFree(d_img);
hipDeviceReset();
return;
}

