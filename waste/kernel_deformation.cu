
#include <hip/hip_runtime.h>
__global__ void kernel_deformation(float *img1, hipTextureObject_t tex_img, float *mx2, float *my2, float *mz2, int nx, int ny, int nz){
    int ix = 16 * blockIdx.x + threadIdx.x;
    int iy = 16 * blockIdx.y + threadIdx.y;
    int iz = 4 * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = iy + ix * ny + iz * nx * ny;
    float xi = iy + 1.0f + my2[id];
    float yi = ix + 1.0f + mx2[id];
    float zi = iz + 1.0f + mz2[id];
    img1[id] = tex3D<float>(tex_img, xi - 0.5f, yi - 0.5f, zi - 0.5f);

    // int id = ix + iy * nx + iz * nx * ny; // index for image
    // int id2 = iy + ix * ny + iz * nx * ny; // index for DVFs
    // float xi = iy + 0.5f + my2[id2];
    // float yi = ix + 0.5f + mx2[id2];
    // float zi = iz + 0.5f + mz2[id2];
    // img1[id2] = tex3D<float>(tex_img, xi, yi, zi);


    // img1[id] = 0.0f;
    // if (xi < 0.5f || xi >= nx - 0.5f || yi < 0.5f || yi >= ny - 0.5f || zi < 0.5f || zi >= nz - 0.5f)
    //     return;
    // if (xi < 0.5f) {xi = 0.5f;}
    // if (xi > nx - 0.5f) {xi = nx - 0.5f;}

    // int ix1, ix2, iy1, iy2, iz1, iz2;
    // float wx1, wx2, wy1, wy2, wz1, wz2;
    // if (xi < 0.5f)
    //     {ix1 = 0; ix2 = 1; wx2 = 0.0f; wx1 = 1.0f;}
    // else{
    //     if (xi >= nx - 0.5f)
    //         {ix1 = nx - 1; ix2 = 1; wx2 = 0.0f; wx1 = 1.0f;}
    //     else
    //         {ix1 = (int)floor(xi - 0.5f); ix2 = ix1 + 1; wx2 = xi - 0.5f - (float)ix1; wx1 = 1.0f - wx2;}
    //     }
    
    // if (yi < 0.5f)
    //     {iy1 = 0; iy2 = 1; wy2 = 0.0f; wy1 = 1.0f;}
    // else{
    //     if (yi >= ny - 0.5f)
    //         {iy1 = ny - 1; iy2 = 1; wy2 = 0.0f; wy1 = 1.0f;}
    //     else
    //         {iy1 = (int)floor(yi - 0.5f); iy2 = iy1 + 1; wy2 = yi - 0.5f - (float)iy1; wy1 = 1.0f - wy2;}
    //     }
    
    // if (zi < 0.5f)
    //     {iz1 = 0; iz2 = 1; wz2 = 0.0f; wz1 = 1.0f;}
    // else{
    //     if (zi >= nz - 0.5f)
    //         {iz1 = nz - 1; iz2 = 1; wz2 = 0.0f; wz1 = 1.0f;}
    //     else           
    //         {iz1 = (int)floor(zi - 0.5f); iz2 = iz1 + 1; wz2 = zi - 0.5f - (float)iz1; wz1 = 1.0f - wz2; }
    //     }
    // ix1 = (int)floor(xi - 0.5f); ix2 = ix1 + 1; wx2 = xi - 0.5f - (float)ix1; wx1 = 1.0f - wx2;
    // iy1 = (int)floor(yi - 0.5f); iy2 = iy1 + 1; wy2 = yi - 0.5f - (float)iy1; wy1 = 1.0f - wy2;
    // iz1 = (int)floor(zi - 0.5f); iz2 = iz1 + 1; wz2 = zi - 0.5f - (float)iz1; wz1 = 1.0f - wz2;
    // img1[id] += img[ix1 + iy1 * nx + iz1 * nx * ny] * wx1 * wy1 * wz1;
    // img1[id] += img[ix1 + iy1 * nx + iz2 * nx * ny] * wx1 * wy1 * wz2;
    // img1[id] += img[ix1 + iy2 * nx + iz1 * nx * ny] * wx1 * wy2 * wz1;
    // img1[id] += img[ix1 + iy2 * nx + iz2 * nx * ny] * wx1 * wy2 * wz2;
    // img1[id] += img[ix2 + iy1 * nx + iz1 * nx * ny] * wx2 * wy1 * wz1;
    // img1[id] += img[ix2 + iy1 * nx + iz2 * nx * ny] * wx2 * wy1 * wz2;
    // img1[id] += img[ix2 + iy2 * nx + iz1 * nx * ny] * wx2 * wy2 * wz1;
    // img1[id] += img[ix2 + iy2 * nx + iz2 * nx * ny] * wx2 * wy2 * wz2;
}


//     int x = blockSize.x * blockIdx.x + threadIdx.x;
//     int y = blockSize.y * blockIdx.y + threadIdx.y;
//     int z = blockSize.z * blockIdx.z + threadIdx.z;
//     if (x >= nx || y >= ny || z >= nz)
//         return;
//     int xi = mx2[x][y][z];
//     int yi = my2[x][y][z];
//     int zi = mz2[x][y][z];

//     singleViewImg1[x][y][z] = tex3D<float>(tex_img, xi-0.5f, yi-0.5f, zi-0.5f);
// }
